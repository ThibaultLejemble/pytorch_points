#include "hip/hip_runtime.h"
#include <torch_points/dummy/dummy.h>
#include <torch_points/common/check.h>

namespace torch_points {

namespace kernel {
    
// 1x1 threads per bloc
__global__ void dummy_print_and_mult10(
    const float* __restrict__ x, 
    float* __restrict__ y,
    int nrows, 
    int ncols)
{
    const int i = blockIdx.x;
    const int j = blockIdx.y;
    if(i < nrows and j < ncols)
    {
        const int k = i * ncols + j; // row-major
        printf("CUDA x[%d,%d] = %f\n", i, j, x[k]);
        y[k] = 10 * x[k];
    }
}

} // namespace kernel


torch::Tensor dummy_cuda(torch::Tensor x)
{
    CHECK_CUDA(x);
    const int nrows = x.size(0);
    const int ncols = x.size(1);
    auto y = torch::Tensor(x);
    // 1x1 threads per bloc
    const dim3 blocs = {uint(nrows), uint(ncols), 1};
    const dim3 threads = {1, 1, 1};
    kernel::dummy_print_and_mult10<<<blocs,threads>>>(
        x.data_ptr<float>(), 
        y.data_ptr<float>(),
        nrows, 
        ncols);
    return y;
}

} // namespace torch_points
